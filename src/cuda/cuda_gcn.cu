#include "hip/hip_runtime.h"
#include "cuda_gcn.cuh"
#include "timer.h"
#include <algorithm>
#include <thrust/transform.h>

using std::max;
using std::max_element;

CUDAGCN::CUDAGCN(GCNParams params, GCNData *input_data) {

    cuda_init_random_state(MAX_THREAD_PER_BLOCK);

    this->params = params;
    data = input_data;
    sp = new CUDASparseIndex(data->feature_index);
    graph = new CUDASparseIndex(data->graph);
    modules.reserve(8);
    variables.reserve(8);

    // dropout
    variables.emplace_back(data->feature_index.indices.size(), false);
    input = &variables.back();
    modules.push_back(new CUDADropout(input, params.dropout));
    
    // sparse matmul
    variables.emplace_back(params.num_nodes * params.hidden_dim);
    CUDAVariable *layer1_var1 = &variables.back();
    variables.emplace_back(params.input_dim * params.hidden_dim, true);
    CUDAVariable *layer1_weight = &variables.back();
    layer1_weight->glorot(params.input_dim, params.hidden_dim);
    modules.push_back(new CUDASparseMatmul(input, layer1_weight, layer1_var1, sp, params.num_nodes, params.input_dim, params.hidden_dim));
    
    // graph sum
    variables.emplace_back(params.num_nodes * params.hidden_dim);
    CUDAVariable *layer1_var2 = &variables.back();
    modules.push_back(new CUDAGraphSum(layer1_var1, layer1_var2, graph, params.hidden_dim));

    // ReLU
    modules.push_back(new CUDAReLU(layer1_var2));

    // dropout
    modules.push_back(new CUDADropout(layer1_var2, params.dropout));

    // dense matmul
    variables.emplace_back(params.num_nodes * params.output_dim);
    CUDAVariable *layer2_var1 = &variables.back();
    variables.emplace_back(params.hidden_dim * params.output_dim, true);
    CUDAVariable *layer2_weight = &variables.back();
    layer2_weight->glorot(params.hidden_dim, params.output_dim);
    modules.push_back(new CUDAMatmul(layer1_var2, layer2_weight, layer2_var1, params.num_nodes, params.hidden_dim, params.output_dim));

    // graph sum
    variables.emplace_back(params.num_nodes * params.output_dim);
    output = &variables.back();
    modules.push_back(new CUDAGraphSum(layer2_var1, output, graph, params.output_dim));

    // cross entropy loss
    CUDA_CHECK(hipMalloc((void**) &truth, params.num_nodes * sizeof(int)));
    modules.push_back(new CUDACrossEntropyLoss(output, truth, &loss, params.output_dim));

    // optimizer
    AdamParams adam_params = AdamParams::get_default();
    adam_params.lr = params.learning_rate;
    adam_params.weight_decay = params.weight_decay;
    optimizer = new CUDAAdam({{layer1_weight, true}, {layer2_weight, false}}, adam_params);

    // other variable
    CUDA_CHECK(hipMalloc((void**) &d_l2_penalty, variables[2].size * sizeof(float)));
}

CUDAGCN::~CUDAGCN() {
    cuda_free_random_state();
    for (auto &m : modules) delete m;
    delete sp;
    delete graph;
    delete optimizer;
    CUDA_CHECK(hipFree(truth));
    CUDA_CHECK(hipFree(d_l2_penalty));
}

void CUDAGCN::set_input() {
    CUDA_CHECK(hipMemcpy(input->data, data->feature_value.data(), input->size * sizeof(float), hipMemcpyHostToDevice));
}

void CUDAGCN::set_truth(int current_split) {
    int *d_data_split, *d_data_label;
    CUDA_CHECK(hipMalloc((void**) &d_data_split, params.num_nodes * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**) &d_data_label, params.num_nodes * sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_data_split, data->split.data(), params.num_nodes * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_data_label, data->label.data(), params.num_nodes * sizeof(int), hipMemcpyHostToDevice));
    dim3 block((params.num_nodes-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    cuda_set_truth_kernel<<<block, thread_in_block>>>(truth, d_data_split, d_data_label, current_split, params.num_nodes);
    CUDA_CHECK(hipFree(d_data_split));
    CUDA_CHECK(hipFree(d_data_label));
}

// TODO: reduction (using thrust?)
float CUDAGCN::get_accuracy() {
    int *cpu_truth = new int[params.num_nodes];
    float *cpu_output = new float[output->size];
    CUDA_CHECK(hipMemcpy(cpu_truth, truth, params.num_nodes * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(cpu_output, output->data, output->size * sizeof(float), hipMemcpyDeviceToHost));

    int wrong = 0, total = 0;
    for(int i = 0; i < params.num_nodes; i++) {
        if(cpu_truth[i] < 0) continue;
        total++;
        float truth_logit = cpu_output[i * params.output_dim + cpu_truth[i]];
        for(int j = 0; j < params.output_dim; j++)
            if (cpu_output[i * params.output_dim + j] > truth_logit) {
                wrong++;
                break;
            }
    }
    delete[] cpu_truth;
    delete[] cpu_output;
    return float(total - wrong) / total;
}

struct square_functor{
    square_functor() {}
    __host__ __device__ float operator()(const float &x) const {
        return x * x;
    }
};
float CUDAGCN::get_l2_penalty() {
    int size = variables[2].size;
    thrust::device_ptr<float> l2_ptr(d_l2_penalty), var2_ptr(variables[2].data);
    thrust::transform(var2_ptr, var2_ptr + size, l2_ptr, square_functor());
    float l2 = thrust::reduce(l2_ptr, l2_ptr + size, (float)0.0, thrust::plus<float>());
    return params.weight_decay * l2 / 2;
}

pair<float, float> CUDAGCN::train_epoch() {
    set_input();
    set_truth(1);
    for (auto m: modules)
        m->forward(true);
    float train_loss = loss + get_l2_penalty();
    float train_acc = get_accuracy();
    for (int i = modules.size() - 1; i >= 0; i--)
        modules[i]->backward();
    optimizer->step();
    return {train_loss, train_acc};
}

pair<float, float> CUDAGCN::eval(int current_split) {
    set_input();
    set_truth(current_split);
    for (auto m: modules)
        m->forward(false);
    float test_loss = loss + get_l2_penalty();
    float test_acc = get_accuracy();
    return {test_loss, test_acc};
}

void CUDAGCN::run() {
    int epoch = 1;

    std::vector<float> loss_history;
    for(; epoch <= params.epochs; epoch++) {
        float train_loss, train_acc, val_loss, val_acc;
        timer_start(TMR_TRAIN);
        std::tie(train_loss, train_acc) = train_epoch();
        std::tie(val_loss, val_acc) = eval(2);
        printf("epoch=%d train_loss=%.5f train_acc=%.5f val_loss=%.5f val_acc=%.5f time=%.5f\n",
            epoch, train_loss, train_acc, val_loss, val_acc, timer_stop(TMR_TRAIN));
        loss_history.push_back(val_loss);
        if(params.early_stopping > 0 && epoch >= params.early_stopping) {
            float recent_loss = 0.0;
            for(int i = epoch - params.early_stopping; i < epoch; i++)
                recent_loss += loss_history[i];
            if (val_loss > recent_loss / params.early_stopping) {
                printf("Early stopping...\n");
                break;
            }
        }
    }
    printf("total training time=%.5f\n", timer_total(TMR_TRAIN));

    float test_loss, test_acc;
    timer_start(TMR_TEST);
    std::tie(test_loss, test_acc) = eval(3);
    printf("test_loss=%.5f test_acc=%.5f time=%.5f\n", test_loss, test_acc, timer_stop(TMR_TEST));
}

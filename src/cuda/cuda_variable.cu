#include "cuda_variable.cuh"

CUDAVariable::CUDAVariable(int size, bool requires_grad) {
    this->requires_grad = requires_grad;
    CUDA_CHECK(hipMalloc((void**) &data, size * sizeof(float)));
    if (requires_grad) {
        CUDA_CHECK(hipMalloc((void**) &grad, size * sizeof(float)));
    }
}

CUDAVariable::~CUDAVariable() {
    CUDA_CHECK(hipFree(data));
    if (requires_grad) CUDA_CHECK(hipFree(grad));
}

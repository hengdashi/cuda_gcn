#include "hip/hip_runtime.h"
#include "cuda_variable.cuh"

CUDAVariable::CUDAVariable(int size, bool requires_grad) {
    this->requires_grad = requires_grad;
    this->size = size;
    CUDA_CHECK(hipMalloc((void**) &data, size * sizeof(float)));
    if (requires_grad) {
        CUDA_CHECK(hipMalloc((void**) &grad, size * sizeof(float)));
    }
}

CUDAVariable::~CUDAVariable() {
    CUDA_CHECK(hipFree(data));
    if (requires_grad) CUDA_CHECK(hipFree(grad));
}

__global__ void cuda_Variable_glorot_kernel(float *data, hiprandState *state, int size, float scale) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
        data[id] = (hiprand_uniform(&state[id % MAX_THREAD_PER_BLOCK]) - 0.5) * scale;
}

void CUDAVariable::glorot(int in_size, int out_size) {
    float range = sqrtf(6.0f / (in_size + out_size)), scale = range * 2;

    dim3 block((size-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    cuda_Variable_glorot_kernel<<<block, thread_in_block>>>(data, devStates, size, scale);
}

void CUDAVariable::zero() {
    CUDA_CHECK(hipMemset(data, 0, size * sizeof(float)));
}

void CUDAVariable::zero_grad() {
    CUDA_CHECK(hipMemset(grad, 0, size * sizeof(float)));
}

void CUDAVariable::print(int col) {
    float cpu_data[size];
    CUDA_CHECK(hipMemcpy(cpu_data, data, size * sizeof(float), hipMemcpyDeviceToHost));
    int count = 0;
    printf("---------DATA----------\n");
    for (int i = 0; i < size; ++i) {
        printf("%.4f ", cpu_data[i]);
        count++;
        if (count % col == 0) printf("\n");
    }
    printf("\n");
}

float CUDAVariable::grad_norm() {
    float norm = 0;
    float *cpu_grad = new float[size];
    CUDA_CHECK(hipMemcpy(cpu_grad, grad, size * sizeof(float), hipMemcpyDeviceToHost));
    for(int i = 0; i < size; ++i)
        norm += cpu_grad[i] * cpu_grad[i];
    delete[] cpu_grad;
    return sqrtf(norm);
}

CUDASparseIndex::CUDASparseIndex(const SparseIndex &sp) {
    indices_size = sp.indices.size();
    indptr_size = sp.indptr.size();

    CUDA_CHECK(hipMalloc((void**) &indices, indices_size * sizeof(int)));
    CUDA_CHECK(hipMemcpy(indices, sp.indices.data(), indices_size * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void**) &indptr, indptr_size * sizeof(int)));
    CUDA_CHECK(hipMemcpy(indptr, sp.indptr.data(), indptr_size * sizeof(int), hipMemcpyHostToDevice));
}

CUDASparseIndex::~CUDASparseIndex() {
    if (indices != nullptr) hipFree(indices);
    if (indptr != nullptr) hipFree(indptr);
}
#include "hip/hip_runtime.h"
#include "cudaacc.cuh"

#define TILE_SIZE 32
#define MAX_THREAD_PER_BLOCK 1024

__global__ void cudaMatMul(
    const float *a, 
    const float *b, 
    float *c, 
    const uint m, 
    const uint n, 
    const uint p) {

    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    int range = (n-1)/TILE_SIZE+1;
    float res = 0;

    for (int i = 0; i < range; i++) {
        if (row < m && i * TILE_SIZE + tx < n)
            tileA[ty][tx] = a[row * n + i * TILE_SIZE + tx];
        else
            tileA[ty][tx] = 0;
        if (col < p && i * TILE_SIZE + ty < n) 
            tileB[ty][tx] = b[(i * TILE_SIZE + ty) * p + col];
        else
            tileB[ty][tx] = 0;

        __syncthreads();
        for (int j = 0; j < TILE_SIZE; j++)
            res += tileA[ty][j] * tileB[j][tx];
        __syncthreads();
    }
    if (row < m && col < p)
        c[row * p + col] = res;

}

void cudaCallMatMulForward(
    const float *a,
    const float *b,
    float *c,
    const uint m, 
    const uint n, 
    const uint p) {
    
    dim3 block((p-1)/TILE_SIZE+1, (m-1)/TILE_SIZE+1, 1);
    dim3 thread_in_block(TILE_SIZE, TILE_SIZE, 1);
    cudaMatMul<<<block, thread_in_block>>>(a, b, c, m, n, p);
}

__global__ void cudaCalcAGrad(
    float *a_grad, 
    const float *b, 
    const float *c_grad, 
    const uint m, 
    const uint n, 
    const uint p) {

    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    __shared__ float tileCGrad[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    int range = (p-1)/TILE_SIZE+1;
    float res = 0;
    for (int i = 0; i < range; i++) {
        if (row < m && i * TILE_SIZE + tx < p)
            tileCGrad[ty][tx] = c_grad[row * p + i * TILE_SIZE + tx];
        else
            tileCGrad[ty][tx] = 0;
        if (col < n && i * TILE_SIZE + ty < p)
            tileB[ty][tx] = b[col * p + i * TILE_SIZE + ty];
        else
            tileB[ty][tx] = 0;
            
        __syncthreads();
        for (int j = 0; j < TILE_SIZE; j++)
            res += tileCGrad[ty][j] * tileB[j][tx];
        __syncthreads();
    }
    if (row < m && col < n)
        a_grad[row * n + col] = res;
}

__global__ void cudaCalcBGrad(
    float *b_grad, 
    const float *a, 
    const float *c_grad, 
    const uint m, 
    const uint n, 
    const uint p) {

    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileCGrad[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    int range = (m-1)/TILE_SIZE+1;
    float res = 0;
    for (int i = 0; i < range; i++) {
        if (row < n && i * TILE_SIZE + tx < m) 
            tileA[ty][tx] = a[(i * TILE_SIZE + tx) * n + row];
        else
            tileA[ty][tx] = 0;
        if (col < p && i * TILE_SIZE + ty < m) 
            tileCGrad[ty][tx] = c_grad[(i * TILE_SIZE + ty) * p + col];
        else
            tileCGrad[ty][tx] = 0;
        
        __syncthreads();
        for (int j = 0; j < TILE_SIZE; j++)
            res += tileA[ty][j] * tileCGrad[j][tx];
        __syncthreads();
    }
    if (row < n && col < p)
        b_grad[row * p + col] = res;
    
}

void cudaCallMatMulBackward(
    const float *a,
    const float *b,
    float *a_grad,
    float *b_grad,
    const float *c_grad,
    const uint m,
    const uint n,
    const uint p) {

    dim3 block_a((n-1)/TILE_SIZE+1, (m-1)/TILE_SIZE+1, 1);
    dim3 block_b((p-1)/TILE_SIZE+1, (n-1)/TILE_SIZE+1, 1);
    dim3 thread_in_block(TILE_SIZE, TILE_SIZE, 1);
    cudaCalcAGrad<<<block_a, thread_in_block>>>(a_grad, b, c_grad, m, n, p);
    cudaCalcBGrad<<<block_b, thread_in_block>>>(b_grad, a, c_grad, m, n, p);
}

hiprandState *devStates;

__global__ void setupRandKernel(hiprandState *state) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1234, id, 0, &state[id]);
}

void cudaCallInitRandomState(const uint size) {
    hipMalloc((void**) &devStates, size * sizeof(hiprandState));
    dim3 block((size-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    setupRandKernel<<<block,thread_in_block>>>(devStates);
}

void cudaCallFreeRandomState() {
    hipFree(devStates);
}

__global__ void cudaDropoutForward(
    float *in, 
    bool *mask,
    hiprandState *state,
    const uint size, 
    const float p, 
    const float scale, 
    const bool useMask) {
    
    float x; 
    bool keep;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        x = hiprand_uniform(&state[id]);
        keep = x >= p;
        in[id] *= keep ? scale : 0;
        if (useMask) mask[id] = keep;
    }
}

void cudaCallDropoutForward(
    float *in,
    bool *mask,
    const uint size,
    const float p,
    const bool useMask) {
    
    float scale = 1 / (1 - p);
    dim3 block((size-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    cudaDropoutForward<<<block, thread_in_block>>>(in, mask, devStates, size, p, scale, useMask);
}

__global__ void cudaDropoutBackward(float *in_grad, const bool *mask, const uint size, const float scale) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
        in_grad[id] *= mask[id] ? scale : 0;
}

void cudaCallDropoutBackward(
    float *in_grad,
    const bool *mask,
    const uint size,
    const float p) {
    
    float scale = 1 / (1 - p);
    dim3 block((size-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    cudaDropoutBackward<<<block, thread_in_block>>>(in_grad, mask, size, scale);
}
#include "hip/hip_runtime.h"
#include "kernels.cuh"

hiprandState *devStates;

// matrix mult
__global__
void cuda_Matmul_forward_kernel(const float *a, const float *b, float *c, const uint m, const uint n, const uint p) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    int range = (n-1) / TILE_SIZE + 1;
    float res = 0;

    for (int i = 0; i < range; i++) {
        if (row < m && i * TILE_SIZE + tx < n)
            tileA[ty][tx] = a[row * n + i * TILE_SIZE + tx];
        else
            tileA[ty][tx] = 0;
        if (col < p && i * TILE_SIZE + ty < n)
            tileB[ty][tx] = b[(i * TILE_SIZE + ty) * p + col];
        else
            tileB[ty][tx] = 0;

        __syncthreads();
        for (int j = 0; j < TILE_SIZE; j++)
            res += tileA[ty][j] * tileB[j][tx];
        __syncthreads();
    }
    if (row < m && col < p)
        c[row * p + col] = res;
}

void cuda_Matmul_forward(Variable *a, Variable *b, Variable *c, int m, int n, int p) {
    float *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a, m * n * sizeof(float));
    hipMemcpy(d_a, a->data.data(), m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_b, n * p * sizeof(float));
    hipMemcpy(d_b, b->data.data(), n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_c, m * p * sizeof(float));

    dim3 block((p-1) / TILE_SIZE + 1, (m-1) / TILE_SIZE + 1, 1);
    dim3 thread_in_block(TILE_SIZE, TILE_SIZE, 1);
    cuda_Matmul_forward_kernel<<<block, thread_in_block>>>(d_a, d_b, d_c, m, n, p);
    hipDeviceSynchronize();

    hipMemcpy(c->data.data(), d_c, m * p * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

__global__
void cuda_Matmul_backward_A_kernel(float *a_grad, const float *b, const float *c_grad, const uint m, const uint n, const uint p) {
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    __shared__ float tileCGrad[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    int range = (p-1) / TILE_SIZE + 1;
    float res = 0;
    for (int i = 0; i < range; i++) {
        if (row < m && i * TILE_SIZE + tx < p)
            tileCGrad[ty][tx] = c_grad[row * p + i * TILE_SIZE + tx];
        else
            tileCGrad[ty][tx] = 0;
        if (col < n && i * TILE_SIZE + ty < p)
            tileB[ty][tx] = b[col * p + i * TILE_SIZE + ty];
        else
            tileB[ty][tx] = 0;

        __syncthreads();

        for (int j = 0; j < TILE_SIZE; j++)
            res += tileCGrad[ty][j] * tileB[j][tx];
        __syncthreads();
    }
    if (row < m && col < n)
        a_grad[row * n + col] = res;
}

__global__
void cuda_Matmul_backward_B_kernel(float *b_grad, const float *a, const float *c_grad, const uint m, const uint n, const uint p) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileCGrad[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    int range = (m-1)/TILE_SIZE+1;
    float res = 0;

    for (int i = 0; i < range; i++) {
        if (row < n && i * TILE_SIZE + tx < m)
            tileA[ty][tx] = a[(i * TILE_SIZE + tx) * n + row];
        else
            tileA[ty][tx] = 0;
        if (col < p && i * TILE_SIZE + ty < m)
            tileCGrad[ty][tx] = c_grad[(i * TILE_SIZE + ty) * p + col];
        else
            tileCGrad[ty][tx] = 0;

        __syncthreads();
        for (int j = 0; j < TILE_SIZE; j++)
            res += tileA[ty][j] * tileCGrad[j][tx];
        __syncthreads();
    }
    if (row < n && col < p)
        b_grad[row * p + col] = res;
}

void cuda_Matmul_backward(Variable *a, Variable *b, Variable *c, int m, int n, int p) {
    float *d_a, *d_b, *d_a_g, *d_b_g, *d_c_g;

    hipMalloc((void**) &d_a, m * n * sizeof(float));
    hipMemcpy(d_a, a->data.data(), m * n * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**) &d_b, n * p * sizeof(float));
    hipMemcpy(d_b, b->data.data(), n * p * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**) &d_a_g, m * n * sizeof(float));
    hipMalloc((void**) &d_b_g, n * p * sizeof(float));
    hipMalloc((void**) &d_c_g, m * p * sizeof(float));
    hipMemcpy(d_c_g, c->grad.data(), m * p * sizeof(float), hipMemcpyHostToDevice);

    dim3 block_a((n-1)/TILE_SIZE+1, (m-1)/TILE_SIZE+1, 1);
    dim3 block_b((p-1)/TILE_SIZE+1, (n-1)/TILE_SIZE+1, 1);
    dim3 thread_in_block(TILE_SIZE, TILE_SIZE, 1);
    cuda_Matmul_backward_A_kernel<<<block_a, thread_in_block>>>(d_a_g, d_b, d_c_g, m, n, p);
    hipDeviceSynchronize();
    cuda_Matmul_backward_B_kernel<<<block_b, thread_in_block>>>(d_b_g, d_a, d_c_g, m, n, p);
    hipDeviceSynchronize();

    hipMemcpy(a->grad.data(), d_a_g, m * n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b->grad.data(), d_b_g, n * p * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_a_g);
    hipFree(d_b_g);
    hipFree(d_c_g);
}


// sparse matmul
__global__
void cuda_SparseMatmul_forward_kernel(float *a_in, float *b_in, float *c_in, int *indptr, int *indices, int p) {
    int i = blockIdx.x;
    int k = blockIdx.y * MAX_THREAD_PER_BLOCK + threadIdx.x;
    
    for (int jj = indptr[i]; jj < indptr[i + 1]; jj++){
        int j = indices[jj];
        c_in[i * p + k] += a_in[jj] * b_in[j * p + k];
    }
}

void cuda_SparseMatmul_forward(Variable *a, Variable *b, Variable *c, SparseIndex *sp, int p) {
    float *a_in, *b_in, *c_in;
    int *d_indptr, *d_indices;

    hipMalloc((void**) &a_in, a->data.size() * sizeof(float));
    hipMalloc((void**) &b_in, b->data.size() * sizeof(float));
    hipMalloc((void**) &c_in, c->data.size() * sizeof(float));
    hipMalloc(&d_indptr, sp->indptr.size() * sizeof(int));
    hipMalloc(&d_indices, sp->indices.size() * sizeof(int));

    hipMemcpy(a_in, a->data.data(), a->data.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_in, b->data.data(), b->data.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_in, c->data.data(), c->data.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_indptr, sp->indptr.data(), sp->indptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, sp->indices.data(), sp->indices.size() * sizeof(int), hipMemcpyHostToDevice);

    if(sp->indptr.size() <= 1) return;

    dim3 gridsize(sp->indptr.size() - 1, 1);
    dim3 blocksize(p);

    if(p > MAX_THREAD_PER_BLOCK) {
        blocksize.x = MAX_THREAD_PER_BLOCK;
        gridsize.y = ceil((float)p / (float)blocksize.x);
    }

    cuda_SparseMatmul_forward_kernel<<<gridsize, blocksize>>>(a_in, b_in, c_in, d_indptr, d_indices, p);

    hipMemcpy(c->data.data(), c_in, c->data.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(a_in);
    hipFree(b_in);
    hipFree(c_in);
    hipFree(d_indptr);
    hipFree(d_indices);
}

__global__
void cuda_SparseMatmul_backward_kernel(float *a_in, float *b_in, float *c_in, int *indptr, int *indices, int p) {
    int i = blockIdx.x;
    int k = blockIdx.y * MAX_THREAD_PER_BLOCK + threadIdx.x;
    
    for (int jj = indptr[i]; jj < indptr[i + 1]; jj++){
        int j = indices[jj];
        b_in[j * p + k] += c_in[i * p + k] * a_in[jj];
    }
}

void cuda_SparseMatmul_backward(Variable *a, Variable *b, Variable *c, SparseIndex *sp, int p) {
    float *a_in, *b_in, *c_in;
    int *d_indptr, *d_indices;

    hipMalloc((void**) &a_in, a->data.size() * sizeof(float));
    hipMalloc((void**) &b_in, b->grad.size() * sizeof(float));
    hipMalloc((void**) &c_in, c->grad.size() * sizeof(float));
    hipMalloc((void**) &d_indptr, sp->indptr.size() * sizeof(int));
    hipMalloc((void**) &d_indices, sp->indices.size() * sizeof(int));

    hipMemcpy(a_in, a->data.data(), a->data.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(b_in, b->grad.data(), b->grad.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_in, c->grad.data(), c->grad.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_indptr, sp->indptr.data(), sp->indptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, sp->indices.data(), sp->indices.size() * sizeof(int), hipMemcpyHostToDevice);

    if(sp->indptr.size() <= 1) return;

    dim3 gridsize(sp->indptr.size() - 1, 1);
    dim3 blocksize(p);

    if(p > MAX_THREAD_PER_BLOCK) {
        blocksize.x = MAX_THREAD_PER_BLOCK;
        gridsize.y = ceil((double)p / (double) blocksize.x);
    }

    cuda_SparseMatmul_backward_kernel<<<gridsize, blocksize>>>(a_in, b_in, c_in, d_indptr, d_indices, p);

    hipMemcpy(b->grad.data(), b_in, b->grad.size() * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(a_in);
    hipFree(b_in);
    hipFree(c_in);
    hipFree(d_indptr);
    hipFree(d_indices);
}


// graph sum
__global__
void cuda_GraphSum_forward_kernel(float *d_in_data, float *d_out_data, int *d_indptr, int *d_indices, int dim, int numNodes) {
    int src = blockIdx.x;
    int j = blockIdx.y * MAX_THREAD_PER_BLOCK + threadIdx.x;

    for (int i = d_indptr[src]; i < d_indptr[src + 1]; i++) {
        int dst = d_indices[i];
        float coef = 1.0 / sqrtf(
                (d_indptr[src + 1] - d_indptr[src]) * (d_indptr[dst + 1] - d_indptr[dst])
        );
        // This only works for undirected graphs. Should be out[dst] += coef * in[src]
        d_out_data[src * dim + j] += coef * d_in_data[dst * dim + j];
    }
    // }
}

void cuda_GraphSum_forward(Variable *in, Variable *out, SparseIndex *graph, int dim) {
    float *d_in_data, *d_out_data;
    int *d_indptr, *d_indices;

    // allocate memory
    hipMalloc((void**) &d_in_data, in->data.size() * sizeof(float));
    hipMalloc((void**) &d_out_data, out->data.size() * sizeof(float));
    hipMalloc((void**) &d_indptr, graph->indptr.size() * sizeof(int));
    hipMalloc((void**) &d_indices, graph->indices.size() * sizeof(int));

    // copy memory from host to device
    hipMemcpy(d_in_data, in->data.data(), in->data.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_indptr, graph->indptr.data(), graph->indptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, graph->indices.data(), graph->indices.size() * sizeof(int), hipMemcpyHostToDevice);

    // kernel
    const int numNodes = graph->indptr.size() - 1;
    dim3 numBlocks(numNodes, 1);
    dim3 threadsPerBlock(dim, 1);

    if(dim > MAX_THREAD_PER_BLOCK) {
        numBlocks.x = MAX_THREAD_PER_BLOCK;
        threadsPerBlock.y = ceil((float)dim / (float)numBlocks.x);
    }

    cuda_GraphSum_forward_kernel<<<numBlocks, threadsPerBlock>>>(d_in_data, d_out_data, d_indptr, d_indices, dim, numNodes);
    hipDeviceSynchronize();

    // copy result back to out
    hipMemcpy(out->data.data(), d_out_data, out->data.size() * sizeof(float), hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_in_data);
    hipFree(d_out_data);
    hipFree(d_indptr);
    hipFree(d_indices);
}

__global__
void cuda_GraphSum_backward_kernel(float *d_in_grad, float *d_out_grad, int *d_indptr, int *d_indices, int dim, int numNodes) {
    int src = blockIdx.x;
    int j = blockIdx.y * MAX_THREAD_PER_BLOCK + threadIdx.x;

    // for (int src = 0; src < numNodes; ++src) {
    for (int i = d_indptr[src]; i < d_indptr[src + 1]; i++) {
        int dst = d_indices[i];
        float coef = 1.0 / sqrtf(
                (d_indptr[src + 1] - d_indptr[src]) * (d_indptr[dst + 1] - d_indptr[dst])
        );
        // This only works for undirected graphs. Should be out[dst] += coef * in[src]
        d_in_grad[src * dim + j] += coef * d_out_grad[dst * dim + j];
    }
    // }
}

void cuda_GraphSum_backward(Variable *in, Variable *out, SparseIndex *graph, int dim) {
    float *d_in_grad, *d_out_grad;
    int *d_indptr, *d_indices;

    // allocate memory
    hipMalloc((void**) &d_in_grad, in->grad.size() * sizeof(float));
    hipMalloc((void**) &d_out_grad, out->grad.size() * sizeof(float));
    hipMalloc((void**) &d_indptr, graph->indptr.size() * sizeof(int));
    hipMalloc((void**) &d_indices, graph->indices.size() * sizeof(int));

    // copy memory from host to device
    hipMemcpy(d_out_grad, out->grad.data(), out->grad.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_indptr, graph->indptr.data(), graph->indptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, graph->indices.data(), graph->indices.size() * sizeof(int), hipMemcpyHostToDevice);

    // kernel
    const int numNodes = graph->indptr.size() - 1;
    dim3 numBlocks(numNodes, 1);
    dim3 threadsPerBlock(dim, 1);

    if(dim > MAX_THREAD_PER_BLOCK) {
        numBlocks.x = MAX_THREAD_PER_BLOCK;
        threadsPerBlock.y = ceil((float)dim / (float)numBlocks.x);
    }

    cuda_GraphSum_backward_kernel<<<numBlocks, threadsPerBlock>>>(d_in_grad, d_out_grad, d_indptr, d_indices, dim, numNodes);
    hipDeviceSynchronize();

    // copy result back to out
    hipMemcpy(in->grad.data(), d_in_grad, in->grad.size() * sizeof(float), hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_in_grad);
    hipFree(d_out_grad);
    hipFree(d_indptr);
    hipFree(d_indices);
}


// cross entropy
__global__ 
void cuda_CrossEntropy_forward_kernel(float* logits_data, float* logits_grad, bool training, int num_classes, int* truth, int* count, float* thread_loss, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= size) return;
    if (truth[i] < 0) {
        count[i] = 0;
        return;
    }
    float *logit = &logits_data[i * num_classes];
    float max_logit = -1e30, sum_exp = 0;
    for (int j = 0; j < num_classes; j++)
        max_logit = fmax(max_logit, logit[j]);
    for (int j = 0; j < num_classes; j++) {
        logit[j] -= max_logit;
        sum_exp += expf(logit[j]);
    }
    if (training) {
        for (int j = 0; j < num_classes; j++) {
            float prob = expf(logit[j]) / sum_exp;
            logits_grad[i * num_classes + j] = prob;
        }
        logits_grad[i * num_classes + truth[i]] -= 1.0;
    }
    count[i] = 1;
    thread_loss[i] = logf(sum_exp) - logit[truth[i]];
}

void cuda_CrossEntropy_forward(Variable *logits, int *truth, float &total_loss, int &count, int num_classes, bool training) {
    // grid + block size
    int grid = 32;
    int block = (logits->data.size()/num_classes+32) / 32;
    // printf("block size: %d\n", block);	
    // printf("logits data size: %ld\n", logits->data.size());	
    // printf("num_classes: %d\n", num_classes);

    // data structures in GPU:
    float* d_logits_data, *d_loss, *d_logits_grad;
    int* d_truth, *d_count;
    int logits_data_size = (int)(logits->data.size())*sizeof(float);
    int logits_grad_size = (int)(logits->grad.size())*sizeof(float);
    int loss_size = (int)(logits->data.size()/num_classes)*sizeof(float);
    int truth_size = (int)(logits->data.size()/num_classes)*sizeof(int);

    // host function variables
    float *logits_data = logits->data.data();
    float *logits_grad = logits->grad.data();

    // hipMalloc
    hipMalloc(&d_logits_data, logits_data_size);
    hipMalloc(&d_logits_grad, logits_grad_size);
    hipMalloc(&d_loss, loss_size);
    hipMalloc(&d_truth, truth_size);
    hipMalloc(&d_count, truth_size);

    // copy data to GPU memory
    hipMemcpy(d_logits_data, logits_data, logits_data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_logits_grad, logits_grad, logits_grad_size, hipMemcpyHostToDevice);
    hipMemcpy(d_truth, truth, truth_size, hipMemcpyHostToDevice);

    // run kernel function
    cuda_CrossEntropy_forward_kernel<<< grid, block >>>(d_logits_data, d_logits_grad, training, num_classes, d_truth, d_count, d_loss, logits->data.size());
    hipDeviceSynchronize();

    // updates logits->data and logits->grad in host function
    hipMemcpy(&(logits->data[0]), d_logits_data, logits_data_size, hipMemcpyDeviceToHost);
    hipMemcpy(&(logits->grad[0]), d_logits_grad, logits_grad_size, hipMemcpyDeviceToHost);

    // accumulate and add count and total_loss variables by thrust::
    thrust::device_ptr<int> count_ptr = thrust::device_pointer_cast(d_count);
    count = thrust::reduce(count_ptr, count_ptr+(logits->data.size()/num_classes), (int)0, thrust::plus<int>());
    thrust::device_ptr<float> loss_ptr = thrust::device_pointer_cast(d_loss);
    total_loss = thrust::reduce(loss_ptr, loss_ptr+(logits->data.size()/num_classes), (float)0.0, thrust::plus<float>());
    hipDeviceSynchronize();

    // free memory
    hipFree(d_logits_data);
    hipFree(d_logits_grad);
    hipFree(d_loss);
    hipFree(d_truth);
    hipFree(d_count);
}


// ReLU
__global__
void cuda_ReLU_forward_kernel(float *d_in_data, bool *d_mask, const long unsigned int datasize, bool training) {
    uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= datasize) return;

    bool keep = d_in_data[i] > 0;
    if (training) d_mask[i] = keep;
    if (!keep) d_in_data[i] = 0;
}

void cuda_ReLU_forward(Variable *in, bool *mask, bool training) {
    float *d_in_data;
    bool *d_mask;
    const long unsigned int datasize = in->data.size();

    hipMalloc(&d_in_data, datasize * sizeof(float));
    hipMalloc(&d_mask, datasize * sizeof(bool));

    hipMemcpy(d_in_data, in->data.data(), datasize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, datasize * sizeof(bool), hipMemcpyHostToDevice);
    // printf("ReLU data size %lu\n", in->data.size());

    const int bsize = 128;
    dim3 numBlocks(bsize, 1);
    dim3 threadsPerBlock(ceil(float(datasize) / bsize), 1);
    cuda_ReLU_forward_kernel<<<numBlocks, threadsPerBlock>>>(d_in_data, d_mask, datasize, training);
    hipDeviceSynchronize();

    hipMemcpy(in->data.data(), d_in_data, datasize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mask, d_mask, datasize * sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(d_in_data);
    hipFree(d_mask);
}

__global__
void cuda_ReLU_backward_kernel(float *d_in_grad, bool *d_mask, long unsigned int datasize) {
    uint i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i >= datasize) return;

    if (!d_mask[i]) d_in_grad[i] = 0;
}

void cuda_ReLU_backward(Variable *in, bool *mask) {
    float *d_in_grad;
    bool *d_mask;
    const long unsigned int datasize = in->data.size();

    hipMalloc(&d_in_grad, datasize * sizeof(float));
    hipMalloc(&d_mask, datasize * sizeof(bool));

    hipMemcpy(d_in_grad, in->grad.data(), datasize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, datasize * sizeof(bool), hipMemcpyHostToDevice);

    const int bsize = 128;
    dim3 numBlocks(bsize, 1);
    dim3 threadsPerBlock(ceil(float(datasize) / bsize), 1);
    cuda_ReLU_backward_kernel<<<numBlocks, threadsPerBlock>>>(d_in_grad, d_mask, datasize);
    hipDeviceSynchronize();

    hipMemcpy(in->grad.data(), d_in_grad, datasize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in_grad);
    hipFree(d_mask);
}


// Dropout
__global__
void cuda_Dropout_forward_kernel(float *in, int *mask, hiprandState *state, const uint size, const float p, const float scale, const bool useMask) {
    float x;
    bool keep;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        x = hiprand_uniform(&state[id]);
        keep = x >= p;
        in[id] *= keep ? scale : 0;
        if (useMask) mask[id] = keep;
    }
}

void cuda_Dropout_forward(Variable *in, int *mask, float p) {
    int size = in->data.size();
    float *d_in;
    int *d_mask;

    hipMalloc((void**) &d_in, size * sizeof(float));
    hipMemcpy(d_in, in->data.data(), size * sizeof(float), hipMemcpyHostToDevice);

    if (mask) hipMalloc((void**) &d_mask, size * sizeof(int));

    float scale = 1 / (1 - p);
    dim3 block((size-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    cuda_Dropout_forward_kernel<<<block, thread_in_block>>>(d_in, d_mask, devStates, size, p, scale, (mask != nullptr));
    hipDeviceSynchronize();

    hipMemcpy(in->data.data(), d_in, size * sizeof(float), hipMemcpyDeviceToHost);
    if (mask) hipMemcpy(mask, d_mask, size * sizeof(int), hipMemcpyDeviceToHost);
}

__global__
void cuda_Dropout_backward_kernel(float *in_grad, const int *mask, const uint size, const float scale) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
        in_grad[id] *= mask[id] ? scale : 0;
}

void cuda_Dropout_backward(Variable *in, int *mask, float p) {
    uint size = in->data.size();
    float *d_in_g;
    int *d_mask;

    hipMalloc((void**) &d_in_g, size * sizeof(float));
    hipMemcpy(d_in_g, in->grad.data(), size * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_mask, size * sizeof(int));
    hipMemcpy(d_mask, mask, size * sizeof(int), hipMemcpyHostToDevice);

    float scale = 1 / (1 - p);
    dim3 block((size-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    cuda_Dropout_backward_kernel<<<block, thread_in_block>>>(d_in_g, d_mask, size, scale);
    hipDeviceSynchronize();

    hipMemcpy(in->grad.data(), d_in_g, size * sizeof(float), hipMemcpyDeviceToHost);
}


// rand state
__global__
void cuda_init_rand_kernel(hiprandState *state) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1234, id, 0, &state[id]);
}

void cuda_init_random_state(const uint size) {
    hipMalloc((void**) &devStates, size * sizeof(hiprandState));
    dim3 block((size-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    cuda_init_rand_kernel<<<block,thread_in_block>>>(devStates);
    hipDeviceSynchronize();
}

void cuda_free_random_state() {
    hipFree(devStates);
}

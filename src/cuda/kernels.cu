#include "hip/hip_runtime.h"
#include "kernels.cuh"

hiprandState *devStates;
#define TILE_SIZE 32
#define MAX_THREAD_PER_BLOCK 1024

// matrix mult
__global__
void cuda_Matmul_forward_kernel(const float *a, const float *b, float *c, const uint m, const uint n, const uint p) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    int range = (n-1) / TILE_SIZE + 1;
    float res = 0;

    for (int i = 0; i < range; i++) {
        if (row < m && i * TILE_SIZE + tx < n)
            tileA[ty][tx] = a[row * n + i * TILE_SIZE + tx];
        else
            tileA[ty][tx] = 0;
        if (col < p && i * TILE_SIZE + ty < n)
            tileB[ty][tx] = b[(i * TILE_SIZE + ty) * p + col];
        else
            tileB[ty][tx] = 0;

        __syncthreads();
        for (int j = 0; j < TILE_SIZE; j++)
            res += tileA[ty][j] * tileB[j][tx];
        __syncthreads();
    }
    if (row < m && col < p)
        c[row * p + col] = res;
}

void cuda_Matmul_forward(Variable *a, Variable *b, Variable *c, int m, int n, int p) {
    float *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a, m * n * sizeof(float));
    hipMemcpy(d_a, a->data.data(), m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_b, n * p * sizeof(float));
    hipMemcpy(d_b, b->data.data(), n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_c, m * p * sizeof(float));

    dim3 block((p-1) / TILE_SIZE + 1, (m-1) / TILE_SIZE + 1, 1);
    dim3 thread_in_block(TILE_SIZE, TILE_SIZE, 1);
    cuda_Matmul_forward_kernel<<<block, thread_in_block>>>(d_a, d_b, d_c, m, n, p);

    hipMemcpy(c->data.data(), d_c, m * p * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

__global__
void cuda_Matmul_backward_A_kernel(float *a_grad, const float *b, const float *c_grad, const uint m, const uint n, const uint p) {
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    __shared__ float tileCGrad[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    int range = (p-1) / TILE_SIZE + 1;
    float res = 0;
    for (int i = 0; i < range; i++) {
        if (row < m && i * TILE_SIZE + tx < p)
            tileCGrad[ty][tx] = c_grad[row * p + i * TILE_SIZE + tx];
        else
            tileCGrad[ty][tx] = 0;
        if (col < n && i * TILE_SIZE + ty < p)
            tileB[ty][tx] = b[col * p + i * TILE_SIZE + ty];
        else
            tileB[ty][tx] = 0;

        __syncthreads();

        for (int j = 0; j < TILE_SIZE; j++)
            res += tileCGrad[ty][j] * tileB[j][tx];
        __syncthreads();
    }
    if (row < m && col < n)
        a_grad[row * n + col] = res;
}

__global__
void cuda_Matmul_backward_B_kernel(float *b_grad, const float *a, const float *c_grad, const uint m, const uint n, const uint p) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileCGrad[TILE_SIZE][TILE_SIZE];
    int bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x, ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;
    int range = (m-1)/TILE_SIZE+1;
    float res = 0;

    for (int i = 0; i < range; i++) {
        if (row < n && i * TILE_SIZE + tx < m)
            tileA[ty][tx] = a[(i * TILE_SIZE + tx) * n + row];
        else
            tileA[ty][tx] = 0;
        if (col < p && i * TILE_SIZE + ty < m)
            tileCGrad[ty][tx] = c_grad[(i * TILE_SIZE + ty) * p + col];
        else
            tileCGrad[ty][tx] = 0;

        __syncthreads();
        for (int j = 0; j < TILE_SIZE; j++)
            res += tileA[ty][j] * tileCGrad[j][tx];
        __syncthreads();
    }
    if (row < n && col < p)
        b_grad[row * p + col] = res;
}

void cuda_Matmul_backward(Variable *a, Variable *b, Variable *c, int m, int n, int p) {
    float *d_a, *d_b, *d_a_g, *d_b_g, *d_c_g;

    hipMalloc((void**) &d_a, m * n * sizeof(float));
    hipMemcpy(d_a, a->data.data(), m * n * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**) &d_b, n * p * sizeof(float));
    hipMemcpy(d_b, b->data.data(), n * p * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**) &d_a_g, m * n * sizeof(float));
    hipMalloc((void**) &d_b_g, n * p * sizeof(float));
    hipMalloc((void**) &d_c_g, m * p * sizeof(float));
    hipMemcpy(d_c_g, c->grad.data(), m * p * sizeof(float), hipMemcpyHostToDevice);

    dim3 block_a((n-1)/TILE_SIZE+1, (m-1)/TILE_SIZE+1, 1);
    dim3 block_b((p-1)/TILE_SIZE+1, (n-1)/TILE_SIZE+1, 1);
    dim3 thread_in_block(TILE_SIZE, TILE_SIZE, 1);
    cuda_Matmul_backward_A_kernel<<<block_a, thread_in_block>>>(d_a_g, d_b, d_c_g, m, n, p);
    cuda_Matmul_backward_B_kernel<<<block_b, thread_in_block>>>(d_b_g, d_a, d_c_g, m, n, p);

    hipMemcpy(a->grad.data(), d_a_g, m * n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b->grad.data(), d_b_g, n * p * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_a_g);
    hipFree(d_b_g);
    hipFree(d_c_g);
}


// graph sum
__global__
void cuda_GraphSum_forward_kernel(float *d_in_data, float *d_out_data, int *d_indptr, int *d_indices, int dim, int numNodes) {
    // printf("graphsum forward loop count: %lu\n", nodecount);
    uint src = (blockIdx.x * blockDim.x) + threadIdx.x;
    // printf("src: %u\n", src);
    if (src >= numNodes) return;
    // printf("src: %d, i: %d, size: %d\n", src, d_indptr[src], d_indptr[src + 1]);
    // for (int src = 0; src < numNodes; ++src) {
    for (int i = d_indptr[src]; i < d_indptr[src + 1]; i++) {
        int dst = d_indices[i];
        float coef = 1.0 / sqrtf(
                (d_indptr[src + 1] - d_indptr[src]) * (d_indptr[dst + 1] - d_indptr[dst])
        );
        // printf("dim: %d\n", dim);
        for (int j = 0; j < dim; j++) {
            // This only works for undirected graphs. Should be out[dst] += coef * in[src]
            d_out_data[src * dim + j] += coef * d_in_data[dst * dim + j];
        }
    }
    // }
}

void cuda_GraphSum_forward(Variable *in, Variable *out, SparseIndex *graph, int dim) {
    float *d_in_data, *d_out_data;
    int *d_indptr, *d_indices;

    // allocate memory
    hipMalloc(&d_in_data, in->data.size() * sizeof(float));
    hipMalloc(&d_out_data, out->data.size() * sizeof(float));
    hipMalloc(&d_indptr, graph->indptr.size() * sizeof(int));
    hipMalloc(&d_indices, graph->indices.size() * sizeof(int));

    // copy memory from host to device
    hipMemcpy(d_in_data, in->data.data(), in->data.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_indptr, graph->indptr.data(), graph->indptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, graph->indices.data(), graph->indices.size() * sizeof(int), hipMemcpyHostToDevice);

    // kernel
    // printf("size of indptr: %lu\n", graph->indptr.size());
    const int numNodes = graph->indptr.size() - 1;
    const int bsize = 32;
    dim3 numBlocks(bsize, 1);
    dim3 threadsPerBlock(ceil(float(numNodes)/bsize), 1);
    // dim3 numBlocks(1, 1);
    // dim3 threadsPerBlock(1, 1);
    cuda_GraphSum_forward_kernel<<<numBlocks, threadsPerBlock>>>(d_in_data, d_out_data, d_indptr, d_indices, dim, numNodes);
    hipDeviceSynchronize();

    // copy result back to out
    hipMemcpy(out->data.data(), d_out_data, out->data.size() * sizeof(float), hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_in_data);
    hipFree(d_out_data);
    hipFree(d_indptr);
    hipFree(d_indices);
}

__global__
void cuda_GraphSum_backward_kernel(float *d_in_grad, float *d_out_grad, int *d_indptr, int *d_indices, int dim, int numNodes) {
    // printf("graphsum backward loop count: %lu\n", nodecount);
    uint src = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (src >= numNodes) return;

    // for (int src = 0; src < numNodes; ++src) {
    for (int i = d_indptr[src]; i < d_indptr[src + 1]; i++) {
        int dst = d_indices[i];
        float coef = 1.0 / sqrtf(
                (d_indptr[src + 1] - d_indptr[src]) * (d_indptr[dst + 1] - d_indptr[dst])
        );
        for (int j = 0; j < dim; j++) {
            // This only works for undirected graphs. Should be out[dst] += coef * in[src]
            d_in_grad[src * dim + j] += coef * d_out_grad[dst * dim + j];
        }
    }
    // }
}

void cuda_GraphSum_backward(Variable *in, Variable *out, SparseIndex *graph, int dim) {
    float *d_in_grad, *d_out_grad;
    int *d_indptr, *d_indices;

    // allocate memory
    hipMalloc(&d_in_grad, in->grad.size() * sizeof(float));
    hipMalloc(&d_out_grad, out->grad.size() * sizeof(float));
    hipMalloc(&d_indptr, graph->indptr.size() * sizeof(int));
    hipMalloc(&d_indices, graph->indices.size() * sizeof(int));

    // copy memory from host to device
    hipMemcpy(d_out_grad, out->grad.data(), out->grad.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_indptr, graph->indptr.data(), graph->indptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, graph->indices.data(), graph->indices.size() * sizeof(int), hipMemcpyHostToDevice);

    // kernel
    const int numNodes = graph->indptr.size() - 1;
    const int bsize = 32;
    dim3 numBlocks(bsize, 1);
    dim3 threadsPerBlock(ceil(float(numNodes) / bsize), 1);
    cuda_GraphSum_backward_kernel<<<numBlocks, threadsPerBlock>>>(d_in_grad, d_out_grad, d_indptr, d_indices, dim, numNodes);
    hipDeviceSynchronize();

    // copy result back to out
    hipMemcpy(in->grad.data(), d_in_grad, in->grad.size() * sizeof(float), hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_in_grad);
    hipFree(d_out_grad);
    hipFree(d_indptr);
    hipFree(d_indices);
}


// cross entropy
__global__ 
void cuda_CrossEntropy_forward_kernel(float* logits_data, float* logits_grad, bool training, int num_classes, int* truth, int* count, float* thread_loss, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= size) return;
    if (truth[i] < 0) {
        count[i] = 0;
        return;
    }
    float *logit = &logits_data[i * num_classes];
    float max_logit = -1e30, sum_exp = 0;
    for (int j = 0; j < num_classes; j++)
        max_logit = fmax(max_logit, logit[j]);
    for (int j = 0; j < num_classes; j++) {
        logit[j] -= max_logit;
        sum_exp += expf(logit[j]);
    }
    if (training) {
        for (int j = 0; j < num_classes; j++) {
            float prob = expf(logit[j]) / sum_exp;
            logits_grad[i * num_classes + j] = prob;
        }
        logits_grad[i * num_classes + truth[i]] -= 1.0;
    }
    count[i] = 1;
    thread_loss[i] = logf(sum_exp) - logit[truth[i]];
}

void cuda_CrossEntropy_forward(Variable *logits, int *truth, float &total_loss, int &count, int num_classes, bool training) {
    // grid + block size
    int grid = 32;
    int block = (logits->data.size()/num_classes+32) / 32;
    // printf("block size: %d\n", block);	
    // printf("logits data size: %ld\n", logits->data.size());	
    // printf("num_classes: %d\n", num_classes);

    // data structures in GPU:
    float* d_logits_data, *d_loss, *d_logits_grad;
    int* d_truth, *d_count;
    int logits_data_size = (int)(logits->data.size())*sizeof(float);
    int logits_grad_size = (int)(logits->grad.size())*sizeof(float);
    int loss_size = (int)(logits->data.size()/num_classes)*sizeof(float);
    int truth_size = (int)(logits->data.size()/num_classes)*sizeof(int);

    // host function variables
    float *logits_data = logits->data.data();
    float *logits_grad = logits->grad.data();

    // hipMalloc
    hipMalloc(&d_logits_data, logits_data_size);
    hipMalloc(&d_logits_grad, logits_grad_size);
    hipMalloc(&d_loss, loss_size);
    hipMalloc(&d_truth, truth_size);
    hipMalloc(&d_count, truth_size);

    // copy data to GPU memory
    hipMemcpy(d_logits_data, logits_data, logits_data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_logits_grad, logits_grad, logits_grad_size, hipMemcpyHostToDevice);
    hipMemcpy(d_truth, truth, truth_size, hipMemcpyHostToDevice);

    // run kernel function
    cuda_CrossEntropy_forward_kernel<<< grid, block >>>(d_logits_data, d_logits_grad, training, num_classes, d_truth, d_count, d_loss, logits->data.size());
    hipDeviceSynchronize();

    // updates logits->data and logits->grad in host function
    hipMemcpy(&(logits->data[0]), d_logits_data, logits_data_size, hipMemcpyDeviceToHost);
    hipMemcpy(&(logits->grad[0]), d_logits_grad, logits_grad_size, hipMemcpyDeviceToHost);

    // accumulate and add count and total_loss variables by thrust::
    thrust::device_ptr<int> count_ptr = thrust::device_pointer_cast(d_count);
    count = thrust::reduce(count_ptr, count_ptr+(logits->data.size()/num_classes), (int)0, thrust::plus<int>());
    thrust::device_ptr<float> loss_ptr = thrust::device_pointer_cast(d_loss);
    total_loss = thrust::reduce(loss_ptr, loss_ptr+(logits->data.size()/num_classes), (float)0.0, thrust::plus<float>());

    // free memory
    hipFree(d_logits_data);
    hipFree(d_logits_grad);
    hipFree(d_loss);
    hipFree(d_truth);
    hipFree(d_count);
}


// ReLU
__global__
void cuda_ReLU_forward_kernel(float *d_in_data, bool *d_mask, const long unsigned int datasize, bool training) {
    uint i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= datasize) return;

    bool keep = d_in_data[i] > 0;
    if (training) d_mask[i] = keep;
    if (!keep) d_in_data[i] = 0;
}

void cuda_ReLU_forward(Variable *in, bool *mask, bool training) {
    float *d_in_data;
    bool *d_mask;
    const long unsigned int datasize = in->data.size();

    hipMalloc(&d_in_data, datasize * sizeof(float));
    hipMalloc(&d_mask, datasize * sizeof(bool));

    hipMemcpy(d_in_data, in->data.data(), datasize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, datasize * sizeof(bool), hipMemcpyHostToDevice);
    // printf("ReLU data size %lu\n", in->data.size());

    const int bsize = 128;
    dim3 numBlocks(bsize, 1);
    dim3 threadsPerBlock(ceil(float(datasize) / bsize), 1);
    cuda_ReLU_forward_kernel<<<numBlocks, threadsPerBlock>>>(d_in_data, d_mask, datasize, training);
    hipDeviceSynchronize();

    hipMemcpy(in->data.data(), d_in_data, datasize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(mask, d_mask, datasize * sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(d_in_data);
    hipFree(d_mask);
}

__global__
void cuda_ReLU_backward_kernel(float *d_in_grad, bool *d_mask, long unsigned int datasize) {
    uint i = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (i >= datasize) return;

    if (!d_mask[i]) d_in_grad[i] = 0;
}

void cuda_ReLU_backward(Variable *in, bool *mask) {
    float *d_in_grad;
    bool *d_mask;
    const long unsigned int datasize = in->data.size();

    hipMalloc(&d_in_grad, datasize * sizeof(float));
    hipMalloc(&d_mask, datasize * sizeof(bool));

    hipMemcpy(d_in_grad, in->grad.data(), datasize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, datasize * sizeof(bool), hipMemcpyHostToDevice);

    const int bsize = 128;
    dim3 numBlocks(bsize, 1);
    dim3 threadsPerBlock(ceil(float(datasize) / bsize), 1);
    cuda_ReLU_backward_kernel<<<numBlocks, threadsPerBlock>>>(d_in_grad, d_mask, datasize);
    hipDeviceSynchronize();

    hipMemcpy(in->grad.data(), d_in_grad, datasize * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_in_grad);
    hipFree(d_mask);
}


// Dropout
__global__
void cuda_Dropout_forward_kernel(float *in, int *mask, hiprandState *state, const uint size, const float p, const float scale, const bool useMask) {
    float x;
    bool keep;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        x = hiprand_uniform(&state[id]);
        keep = x >= p;
        in[id] *= keep ? scale : 0;
        if (useMask) mask[id] = keep;
    }
}

void cuda_Dropout_forward(Variable *in, int *mask, float p) {
    int size = in->data.size();
    float *d_in;
    int *d_mask;

    hipMalloc((void**) &d_in, size * sizeof(float));
    hipMemcpy(d_in, in->data.data(), size * sizeof(float), hipMemcpyHostToDevice);

    if (mask) hipMalloc((void**) &d_mask, size * sizeof(int));

    float scale = 1 / (1 - p);
    dim3 block((size-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    cuda_Dropout_forward_kernel<<<block, thread_in_block>>>(d_in, d_mask, devStates, size, p, scale, (mask != nullptr));

    hipMemcpy(in->data.data(), d_in, size * sizeof(float), hipMemcpyDeviceToHost);
    if (mask) hipMemcpy(mask, d_mask, size * sizeof(int), hipMemcpyDeviceToHost);
}

__global__
void cuda_Dropout_backward_kernel(float *in_grad, const int *mask, const uint size, const float scale) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size)
        in_grad[id] *= mask[id] ? scale : 0;
}

void cuda_Dropout_backward(Variable *in, int *mask, float p) {
    uint size = in->data.size();
    float *d_in_g;
    int *d_mask;

    hipMalloc((void**) &d_in_g, size * sizeof(float));
    hipMemcpy(d_in_g, in->grad.data(), size * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc((void**) &d_mask, size * sizeof(int));
    hipMemcpy(d_mask, mask, size * sizeof(int), hipMemcpyHostToDevice);

    float scale = 1 / (1 - p);
    dim3 block((size-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    cuda_Dropout_backward_kernel<<<block, thread_in_block>>>(d_in_g, d_mask, size, scale);

    hipMemcpy(in->grad.data(), d_in_g, size * sizeof(float), hipMemcpyDeviceToHost);
}


// rand state
__global__
void cuda_init_rand_kernel(hiprandState *state) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(1234, id, 0, &state[id]);
}

void cuda_init_random_state(const uint size) {
    hipMalloc((void**) &devStates, size * sizeof(hiprandState));
    dim3 block((size-1)/MAX_THREAD_PER_BLOCK + 1, 1, 1);
    dim3 thread_in_block(MAX_THREAD_PER_BLOCK, 1, 1);
    cuda_init_rand_kernel<<<block,thread_in_block>>>(devStates);
}

void cuda_free_random_state() {
    hipFree(devStates);
}
